
#include <hip/hip_runtime.h>
/*
 * Copyright 2015 Stanford University and NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

__global__ void
__launch_bounds__(64,1)
different_test(void)
{
  int wid = threadIdx.x >> 5;
  if (wid == 0)
    asm volatile("bar.sync 0, 96;" : : : "memory");
  else if (wid == 1)
    asm volatile("bar.sync 0, 128;" : : : "memory");
}

